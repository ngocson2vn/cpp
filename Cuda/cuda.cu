#include "hip/hip_runtime.h"

// Kernel - Adding two matrices MatA and MatB
__global__ void cuda_mat_add(int* d_MatA, int* d_MatB, int* d_MatC, const int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < N && j < N) d_MatC[N * i + j] = d_MatA[N * i + j] + d_MatB[N * i + j];
}

int* mat_add(int* MatA, int* MatB, const int N) {
  // Create a stream
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // Transfer data from host to device
  int* d_MatA;
  hipMalloc((void**)&d_MatA, N * N * sizeof(int));
  hipMemcpyAsync(d_MatA, MatA, N * N * sizeof(int), hipMemcpyHostToDevice, stream);

  int* d_MatB;
  hipMalloc((void**)&d_MatB, N * N * sizeof(int));
  hipMemcpyAsync(d_MatB, MatB, N * N * sizeof(int), hipMemcpyHostToDevice, stream);

  int* d_MatC;
  hipMalloc((void**)&d_MatC, N * N * sizeof(int));

  // Launch MatAdd kernel
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
  cuda_mat_add<<<numBlocks, threadsPerBlock, 0, stream>>>(d_MatA, d_MatB, d_MatC, N);

  // Transfer results from device to host
  int* MatC = (int*)calloc(N * N, sizeof(int));
  hipMemcpyAsync(MatC, d_MatC, N * N * sizeof(int), hipMemcpyDeviceToHost, stream);

  hipDeviceSynchronize();

  // Free device memory
  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);

  return MatC;
}
